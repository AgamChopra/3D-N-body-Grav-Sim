﻿
#include "hip/hip_runtime.h"
#include ""
#include "utils.h"

#include <stdio.h>

int main() {
    double A[N][N] = { {1.2, 9.2, 4.3}, {4.84, 0.25, 7.126}, {327.7, 568.1, 9.452} };
    double B[N][N] = { {10.5, 20.45534, 30.45}, {40.4566, 50.634, 60.45}, {70.563, 80.34, 90.54} };
    double C[N][N];

    double* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(double));
    hipMalloc((void**)&d_B, N * N * sizeof(double));
    hipMalloc((void**)&d_C, N * N * sizeof(double));

    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(double), hipMemcpyHostToDevice);
    dim3 blockSize(N, N);
    dim3 gridSize(1, 1);

    // Matrix Addition
    matrixAddition << <gridSize, blockSize >> > (d_A, d_B, d_C);
    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Matrix Addition: " << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Matrix Subtraction
    matrixSubtraction << <gridSize, blockSize >> > (d_A, d_B, d_C);
    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Matrix Subtraction: " << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Matrix Division
    matrixDivision << <gridSize, blockSize >> > (d_A, d_B, d_C);
    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Matrix Division: " << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    // Matrix Multiplication
    matrixMultiplication << <gridSize, blockSize >> > (d_A, d_B, d_C);
    hipMemcpy(C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);
    std::cout << "Matrix Multiplication: " << std::endl;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            std::cout << C[i][j] << " ";
        }
        std::cout << std::endl;
    }

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}